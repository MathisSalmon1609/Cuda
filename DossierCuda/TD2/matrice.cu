
#include <hip/hip_runtime.h>
#include<iostream>

__global__ void fill(int * m, std::size_t w ,  std::size_t h)
{
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    auto idy = blockIdx.y * blockDim.y + threadIdx.y;


    if( idx < w && idy <h )
    {
        m [ idy * w + idx ] = idy * w + idx;
    }

}

int main() {

   std::size_t w =10;
   std::size_t h =10;
   std::size_t size =w*h;

   int * m_h = nullptr;
   int * m_d = nullptr;
    hipHostMalloc(&m_h, size * sizeof(int), hipHostMallocDefault);
    hipMalloc( &m_d, size * sizeof(int));
    dim3 block (32 , 32);
    dim3 grid ((w-1) / block.x +1, (h-1)/block.y +1);
    fill<<<grid, block >>>(m_d, w , h );
    hipMemcpy ( m_h , m_d, size * sizeof (int) , hipMemcpyDeviceToHost);

    for (std::size_t j = 0; j < h ;++j )
    {
        for (std::size_t i =0 ; i<w ; ++i)
        {
         std::cout << m_h[j*w +i]<< ' ';
        }
        std::cout << std:: endl;
    }

    hipFree(m_d);
    hipHostFree(m_h);


    return 0;
}
