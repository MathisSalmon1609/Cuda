
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


__global__ void vecadd( int * v0, int * v1, std::size_t size )
{
  auto tid = threadIdx.x;
  v0[ tid ] += v1[ tid ];
}


int main()
{
  hipError_t err;
  
  std::size_t const size = 100;
  std::size_t const sizeb = size * sizeof( int );
  
  std::vector< int > v0( size );
  std::vector< int > v1( size );

  /*
  int * v0_h = nullptr;
  int * v1_h = nullptr;
  */
  
  for( std::size_t i = 0 ; i < size ; ++i )
  {
    v0[ i ] = v1[ i ] = i;
  }
  
  int * v0_d = nullptr;
  int * v1_d = nullptr;
  
  hipHostRegister( v0.data(), sizeb, hipHostRegisterDefault );
  hipHostRegister( v1.data(), sizeb, hipHostRegisterDefault );

  /*
  err = cudaMallocHost( &v0_h, sizeb );
  if( err != cudaSuccess ) { std::cerr << "Error" << std::endl; }
  err = cudaMallocHost( &v1_h, sizeb);
  if( err != cudaSuccess ) { std::cerr << "Error" << std::endl; }
  */
  /*
  for( std::size_t i = 0 ; i < size ; ++i )
  {
    v0_h[ i ] = 5;
    v1_h[ i ] = 5;
  }
  */
  
  err = hipMalloc( &v0_d, sizeb );
  if( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
  err = hipMalloc( &v1_d, sizeb );
  if( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
  
  hipStream_t streams[ 2 ];

  for( std::size_t i = 0 ; i < 2 ; ++i )
  {
    hipStreamCreate( &streams[ i ] );
  }
  for( std::size_t i = 0 ; i < 2 ; ++i )
  {
    err = hipMemcpyAsync( v0_d + i*size/2, v0.data() + i*size/2, sizeb/2, hipMemcpyHostToDevice, streams[ i ] );
    if( err != hipSuccess ) { std::cerr << "Error 3" << std::endl; }
  }

  for( std::size_t i = 0 ; i < 2 ; ++i )
  {
    vecadd<<< 1, 50, 0, streams[ i ] >>>( v0.data() + i*size/2, v1.data() + i*size/2, size/2 );
    err = hipGetLastError();
    if( err != hipSuccess ) { std::cerr << "Error 3.5" << std::endl; }
  }

  for( std::size_t i = 0 ; i < 2 ; ++i )
  {
    err = hipMemcpyAsync( v0.data() + i*size/2, v0_d + i*size/2, sizeb/2, hipMemcpyDeviceToHost, streams[ i ] );
    if( err != hipSuccess ) { std::cerr << "Error 4" << std::endl; }
  }
  hipDeviceSynchronize( );

  for( std::size_t i = 0 ; i < 2 ; ++i )
  {  
    hipStreamDestroy( streams[ i ] );
  }

  for( auto x: v0 )
  {
    std::cout << x << std::endl;
  }
  
  return 0;
}