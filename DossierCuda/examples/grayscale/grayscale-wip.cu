#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void grayscale( unsigned char * rgb, unsigned char * g, std::size_t cols, std::size_t rows ) {
  auto i = ???;
  auto j = ???;
  if( i < cols && j < rows ) {
    ???;
  }
}

int main()
{
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );

  auto rgb = m_in.data;

  auto rows = m_in.rows;
  auto cols = m_in.cols;

  std::vector< unsigned char > g( rows * cols ); // image de sortie.

  cv::Mat m_out( rows, cols, CV_8UC1, g.data() );

  unsigned char * rgb_d;
  unsigned char * g_d;

  hipMalloc( ??? ); // allocation pour l'image d'entrée sur le device.
  hipMalloc( ??? ); // allocation pour l'image de sortie sur le device.

  hipMemcpy( ??? ); // copie de l'image d'entrée vers le device.

  dim3 t( 32, 32 );
  dim3 b( ( cols - 1) / t.x + 1 , ( rows - 1 ) / t.y + 1 );
  grayscale<<< b, t >>>( ??? );

  hipMemcpy( ??? ); // récupération de l'image en niveaux de gris sur l'hôte.

  cv::imwrite( "out.jpg", m_out ); // sauvegarde de l'image.

  hipFree( rgb_d );
  hipFree( g_d);

  return 0;
}
