
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

__global__ void matadd( int * m0, int * m1, std::size_t w, std::size_t h )
{
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;
  if( i < w && j < h )
    m0[ i * w + j ] +=  m1[ i * w + j ];// i * w + j;
}

int main() {
  std::vector< int > v0_h( 10000 );
  std::vector< int > v1_h( 10000 );  
  for( std::size_t i = 0 ; i < v0_h.size(); ++i ) {
    v0_h[ i ] = v1_h[ i ] = i;
  }
  int * v0_d = nullptr;
  int * v1_d = nullptr;
  hipMalloc( &v0_d, v0_h.size() * sizeof( int ) );
  hipMalloc( &v1_d, v0_h.size() * sizeof( int ) );
  hipMemcpy( v0_d, v0_h.data(), v0_h.size() * sizeof( int ), hipMemcpyHostToDevice );
  hipMemcpy( v1_d, v1_h.data(), v0_h.size() * sizeof( int ), hipMemcpyHostToDevice );
  dim3 t( 32, 32 );
  dim3 b( 4, 4 );

  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );
  
  hipEventRecord( start );
  
  matadd<<< b, t >>>( v0_d, v1_d, 100, 100 );

  hipEventRecord( stop );

  hipEventSynchronize( stop );

  float elapsedTime;
  hipEventElapsedTime( & elapsedTime, start, stop );
  std::cout << elapsedTime << std::endl;
  hipEventDestroy( start );
  hipEventDestroy( stop );
  auto err = hipGetLastError();

  hipMemcpy( v0_h.data(), v0_d, v0_h.size() * sizeof( int ), hipMemcpyDeviceToHost );
  //for( auto const i: v0_h ) { std::cout << i << std::endl; }
  hipFree( v0_d );
  hipFree( v1_d );
  return 0;
}
