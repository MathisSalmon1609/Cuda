
#include <hip/hip_runtime.h>
#include<iostream>

__global__ void fill(int * v, std::size_t size)
{
  auto id = blockIdx.x * blockDim.x + threadIdx.x;
  
  if( id < size)
  {
      v [ id ] = id;
  }

}

int main() {

  std::size_t size = 2048;
  int * v_h = nullptr;
  int * v_d = nullptr;

   hipHostMalloc( &v_h, size * sizeof(int), hipHostMallocDefault);
	//ou :  v_h = (int * )malloc(size* sizeof(int));
	//ou : v_h = new int [size];

  hipHostMalloc( &v_d, size * sizeof(int), hipHostMallocDefault);
  dim3 block = 1024;
  dim3 grid = (size -1) / block.x +1;
  fill<<< grid , block >>>( v_d, size );

  hipMemcpy( v_h, v_d, size * sizeof(int), hipMemcpyDeviceToHost);
  
  for ( std::size_t i = 0 ; i < size ; ++i ) {
    std::cout << v_h[i] << std::endl;
  }

  return 0;
}
