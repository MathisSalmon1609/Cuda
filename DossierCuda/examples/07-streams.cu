
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


__global__ void vecadd( int * v0, int * v1, std::size_t size )
{
  auto tid = threadIdx.x;
  v0[ tid ] += v1[ tid ];
}


int main()
{
  hipError_t err;
  
  std::size_t const size = 100;
  std::size_t const sizeb = size * sizeof( int );
  
  int * v0_h = nullptr;
  int * v1_h = nullptr;
  
  int * v0_d = nullptr;
  int * v1_d = nullptr;
  
  err = hipHostMalloc( &v0_h, sizeb , hipHostMallocDefault);
  if( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
  err = hipHostMalloc( &v1_h, sizeb , hipHostMallocDefault);
  if( err != hipSuccess ) { std::cerr << "Error" << std::endl; }

  for( std::size_t i = 0 ; i < size ; ++i )
  {
    v0_h[ i ] = v1_h[ i ] = i;
  }

  err = hipMalloc( &v0_d, sizeb );
  if( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
  err = hipMalloc( &v1_d, sizeb );
  if( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
  
  hipStream_t streams[ 2 ];

  for( std::size_t i = 0 ; i < 2 ; ++i )
  {
    hipStreamCreate( &streams[ i ] );
  }

  for( std::size_t i = 0 ; i < 2 ; ++i )
  {
    hipMemcpyAsync( v0_d + i*size/2, v0_h + i*size/2, sizeb/2, hipMemcpyHostToDevice, streams[ i ] );
    hipMemcpyAsync( v1_d + i*size/2, v1_h + i*size/2, sizeb/2, hipMemcpyHostToDevice, streams[ i ] );
  }

  for( std::size_t i = 0 ; i < 2 ; ++i )
  {
    vecadd<<< 1, size/2, 0, streams[ i ] >>>( v0_d + i*size/2, v1_d + i*size/2, size/2 );
  }
  /*
  cudaDeviceSynchronize();
  err = cudaGetLastError();
  if( err != cudaSuccess )
  {
     std::cout << cudaGetErrorString( err ) << std::endl;
  }
*/

  for( std::size_t i = 0 ; i < 2 ; ++i )
  {
    hipMemcpyAsync( v0_h + i*size/2, v0_d + i*size/2, sizeb/2, hipMemcpyDeviceToHost, streams[ i ] );
  }

  hipDeviceSynchronize();
  /*
  err = cudaGetLastError();
  if( err != cudaSuccess )
  {
     std::cout << cudaGetErrorString( err ) << std::endl;
  }
*/
  for( std::size_t i = 0 ; i < 2 ; ++i )
  {  
    hipStreamDestroy( streams[ i ] );
  }

  for( std::size_t i = 0 ; i < size ; ++i )
  {
    std::cout << v0_h[ i ] << std::endl;
  }

  hipFree( v0_d );
  hipFree( v1_d );

  hipHostFree( v0_h );
  hipHostFree( v1_h );

  return 0;
}