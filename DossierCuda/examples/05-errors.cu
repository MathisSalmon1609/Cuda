
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


__global__ void fill( int * v, std::size_t size )
{
  auto tid = threadIdx.x;
  v[ tid ] = tid;
}


int main()
{
  std::vector< int > v( 100 );

  int * v_d = nullptr;

  hipMalloc( &v_d, v.size() * sizeof( int ) );

  fill<<< 1, 1025 >>>( v_d, v.size() );

  hipDeviceSynchronize();
  auto err2 = hipGetLastError();
  if(err2!= hipSuccess)
  {
    std::cout<<hipGetErrorString(err2);
  }


  // Récupération du code erreur du kernel en cas de plantage.
  hipDeviceSynchronize(); // Attente de la fin d'exécution du kernel.
  hipError_t err = hipGetLastError();
  if( err != hipSuccess )
  {
    std::cerr << hipGetErrorString( err ); // récupération du message associé au code erreur.
  }

  // Récupération du code erreur pour les fonctions CUDA synchrones.
  err = hipMemcpy( v.data(), v_d, v.size() * sizeof( int ), hipMemcpyDeviceToHost );
  if( err != hipSuccess )
  {
    std::cerr << hipGetErrorString( err ); // récupération du message associé au code erreur.
  }

  for( auto x: v )
  {
    std::cout << x << std::endl;
  }

  return 0;
}