
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


__global__ void fill( int * v, std::size_t size )
{
  auto tid = threadIdx.x;
  v[ tid ] = tid;
}


int main()
{
  std::vector< int > v( 100 );

  int * v_d = nullptr;

  hipMalloc( &v_d, v.size() * sizeof( int ) );

  fill<<< 1, 100 >>>( v_d, v.size() );

  hipMemcpy( v.data(), v_d, v.size() * sizeof( int ), hipMemcpyDeviceToHost );

  for( auto x: v )
  {
    std::cout << x << std::endl;
  }

  return 0;
}